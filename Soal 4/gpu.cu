#include <hip/hip_runtime.h>
#include <cstdint>


// Pewarnaan
__device__ uchar4 mapIterationToColor(int n, int max_iter) {
    if (n == max_iter) return make_uchar4(0, 0, 0, 255);
    double t = static_cast<double>(n) / static_cast<double>(max_iter);
    uint8_t r = static_cast<uint8_t>(9 * (1 - t) * t * t * t * 255);
    uint8_t g = static_cast<uint8_t>(15 * (1 - t) * (1 - t) * t * t * 255);
    uint8_t b = static_cast<uint8_t>(8.5 * (1 - t) * (1 - t) * (1 - t) * t * 255);
    return make_uchar4(r, g, b, 255);
}

// Kernel Mandelbrot dengan parameter view
__global__ void mandelbrotKernel(uchar4* pixels, int width, int height, int max_iter, double viewX, double viewY, double zoom) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px >= width || py >= height) return;

    double x0 = (static_cast<double>(px) - width / 2.0) * zoom / width + viewX;
    double y0 = (static_cast<double>(py) - height / 2.0) * zoom / width + viewY;
    
    double x = 0.0, y = 0.0;
    int iteration = 0;
    while (x * x + y * y <= 4.0 && iteration < max_iter) {
        double xtemp = x * x - y * y + x0;
        y = 2.0 * x * y + y0;
        x = xtemp;
        iteration++;
    }
    
    pixels[py * width + px] = mapIterationToColor(iteration, max_iter);
}

// Kernel Julia
__global__ void juliaKernel(uchar4* pixels, int width, int height, int max_iter, double viewX, double viewY, double zoom, double cX, double cY) {
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px >= width || py >= height) return;

    double zx = (static_cast<double>(px) - width / 2.0) * zoom / width + viewX;
    double zy = (static_cast<double>(py) - height / 2.0) * zoom / width + viewY;
    
    int iteration = 0;
    while (zx * zx + zy * zy <= 4.0 && iteration < max_iter) {
        double xtemp = zx * zx - zy * zy + cX;
        zy = 2.0 * zx * zy + cY;
        zx = xtemp;
        iteration++;
    }

    pixels[py * width + px] = mapIterationToColor(iteration, max_iter);
}

// Kernel Mandelbrot
extern "C" void launchMandelbrotKernel(uchar4* pixels, int width, int height, int max_iter, double viewX, double viewY, double zoom) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks( (width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (height + threadsPerBlock.y - 1) / threadsPerBlock.y );
    mandelbrotKernel<<<numBlocks, threadsPerBlock>>>(pixels, width, height, max_iter, viewX, viewY, zoom);
}

// Wrapper untuk kernel Julia
extern "C" void launchJuliaKernel(uchar4* pixels, int width, int height, int max_iter, double viewX, double viewY, double zoom, double cX, double cY) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks( (width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                    (height + threadsPerBlock.y - 1) / threadsPerBlock.y );
    juliaKernel<<<numBlocks, threadsPerBlock>>>(pixels, width, height, max_iter, viewX, viewY, zoom, cX, cY);
}